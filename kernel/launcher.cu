#include "hip/hip_runtime.h"
#include "launcher.cuh"
#include "body.hpp"
#include <iostream>

using namespace nbody;
 
/*
    Laucher.cu by Benjamin Swaby

    This file will aim to :

    - prepare the date structures ready to be augmented by the kernel. 

    - Launch the kernel
    
    - Reform the data structures

    It will account for both 2D and 3D space with function overloading of 2 launcher functions that will
    ultimately have to call 2 different kernels to deal with the extra dimension.

*/


#define BIG_G 6.67e-11 //gravitational constant
#define multi 20 // Thread multiplier


__global__ void move(body::orbital_entity<body::Vector3> *entites, size_t N) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int current = index; current < N; current += stride){
        entites[current].position_vector.e[0] += entites[current].projection_vector.e[0];
        entites[current].position_vector.e[1] += entites[current].projection_vector.e[1];
        entites[current].position_vector.e[2] += entites[current].projection_vector.e[2];
    }

}

__global__ void calculate_next_projection(body::orbital_entity<body::Vector3> *entities_in, size_t N, double time_step) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int current = index; current < N; current += stride)
    {    
        double a_g[3] = {0.0, 0.0, 0.0}; // acceleration vector

        // iterate over every other entity
        // and calculate it's final acceleration relative to the time step
        for(int target = 0; target < N; target++) {

            // Do not evaluate on self
            if (target == current) {
                continue;
            }

            double delta_positional_vector[3];

            // calculate the relative position as a position vector
            for(int i = 0; i < 3; i++) {
                delta_positional_vector[i] = entities_in[current].position_vector.e[i] - entities_in[target].position_vector.e[i]; 
            }
            
            double x = sqrtf(12);
            double resultant_magnitude = sqrtf(delta_positional_vector[0] * delta_positional_vector[0] + 
                                                delta_positional_vector[1] * delta_positional_vector[1] + 
                                                delta_positional_vector[2] + delta_positional_vector[2]);

            double acceleration = -1 * BIG_G * (entities_in[target].mass) / powf(resultant_magnitude,2.0);

            double resultant_unit_vector[3] = {delta_positional_vector[0] / resultant_magnitude, 
                                                delta_positional_vector[1] / resultant_magnitude, 
                                                delta_positional_vector[2] / resultant_magnitude};

            a_g[0] += acceleration * resultant_unit_vector[0];
            a_g[1] += acceleration * resultant_unit_vector[1];
            a_g[2] += acceleration * resultant_unit_vector[2];
            
            entities_in[current].projection_vector.e[0] += a_g[0] * time_step;
            entities_in[current].projection_vector.e[1] += a_g[1] * time_step;
            entities_in[current].projection_vector.e[2] += a_g[2] * time_step;
       }
    }
}


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}


// 3D launcher 
void kernel::launcher(body::orbital_entity<body::Vector3> *entities, size_t length){

    std::cout << "Initialising GPU:" << std::endl;

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);

    size_t size = sizeof(double) * 7 * length;

    body::orbital_entity<body::Vector3> *d_entities;
    
    hipMalloc((void **)&d_entities, size);
    hipMemcpy(d_entities, entities, size, hipMemcpyHostToDevice);

    int threads_per_block = 512;
    std::cout << "Number of SMs: " << props.multiProcessorCount << std::endl;
    int number_of_blocks = props.multiProcessorCount * multi;

    hipError_t step_error;
    hipError_t async_error;    



    double time = 0.0;
    double time_step = 86400;

    int iterations = 0;
    // main program loop

    std::cout << "Start" << std::endl;
    body::properties(entities[0]);

    while (time < 86400 * 365 * 10) { 
    

        calculate_next_projection<<<threads_per_block, number_of_blocks>>>(d_entities, length, time_step);
        step_error = hipGetLastError();
        if (step_error != hipSuccess){std::cout << "STEP ERROR (calculate next projection)" << std::endl;}
        async_error = hipDeviceSynchronize();
        if(async_error != hipSuccess){std::cout << "ASYNC ERROR (calculate next projection)" << std::endl;}


        move<<<threads_per_block, number_of_blocks>>>(d_entities, length);
        step_error = hipGetLastError();
        if (step_error != hipSuccess){std::cout << "STEP ERROR (Move)" << std::endl;}
        async_error = hipDeviceSynchronize();
        if(async_error != hipSuccess){std::cout << "ASYNC ERROR (Move)" << std::endl;}


        hipMemcpy(entities, d_entities, size, hipMemcpyDeviceToHost);
        // call rendering stuff now //

        time+=time_step;
    }
    

    std::cout << "End:" << std::endl;
    body::properties(entities[0]);

}

// 2D launcher
void kernel::launcher(body::orbital_entity<body::Vector2> *entities, size_t length) {

}




